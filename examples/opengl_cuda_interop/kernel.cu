#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>


//// CUDA kernel to set each pixel to a random color
//__global__ void kernel(uchar4* devPtr, int width, int height, int tick) {
//    int x = blockIdx.x * blockDim.x + threadIdx.x;
//    int y = blockIdx.y * blockDim.y + threadIdx.y;
//
//    if (x < width && y < height) {
//        int offset = y * width + x;
//
//        // Simple random number generator based on tick, x, and y
//        unsigned int seed = (tick + x * 1973 + y * 9277) & 0x7FFFFFFF;
//        devPtr[offset].x = (seed % 256);  // Red
//        devPtr[offset].y = ((seed >> 8) % 256);  // Green
//        devPtr[offset].z = ((seed >> 16) % 256);  // Blue
//        devPtr[offset].w = 255;  // Alpha
//    }
//}
//
//void launchKernel(uchar4* devPtr, int width, int height, int tick) {
//    dim3 blockSize(16, 16);
//    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
//    kernel<<<gridSize, blockSize >>> (devPtr, width, height, tick);
//    hipDeviceSynchronize();
//}

struct QuadTreeNode {
    int x, y;
    int size;
    bool isLeaf;
};

// CUDA kernel to draw the Quadtree
__global__ void drawQuadtree(uchar4* devPtr, int width, int height, QuadTreeNode* nodes, int nodeCount) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int offset = y * width + x;

        // Initialize the pixel as black
        devPtr[offset].x = 0;
        devPtr[offset].y = 0;
        devPtr[offset].z = 0;
        devPtr[offset].w = 255;

        // Check each node to see if the pixel lies on a quadtree boundary
        for (int i = 0; i < nodeCount; i++) {
            QuadTreeNode node = nodes[i];

            if (node.isLeaf) {
                // Draw the boundary of the node
                if (x == node.x || x == node.x + node.size - 1 || y == node.y || y == node.y + node.size - 1) {
                    devPtr[offset].x = 255;  // Red color for boundaries
                    devPtr[offset].y = 255;  // Green color for boundaries
                    devPtr[offset].z = 255;  // Blue color for boundaries
                }
            }
        }
    }
}

// CUDA kernel to generate a randomized Quadtree
__global__ void generateRandomQuadtree(QuadTreeNode* nodes, int width, int height, int maxDepth, unsigned long long seed) {
    int nodeIndex = 0;
    hiprandState state;
    hiprand_init(seed, 0, 0, &state);

    __shared__ QuadTreeNode localNodes[1024];
    __shared__ int localIndex;

    if (threadIdx.x == 0) {
        localIndex = 0;
    }
    __syncthreads();

    int x = hiprand(&state) % width;
    int y = hiprand(&state) % height;
    int size = (width < height ? width : height) / 4; // Initial size

    for (int d = 0; d < maxDepth; ++d) {
        if (localIndex < 1024) {
            localNodes[localIndex] = { x, y, size, true };
            ++localIndex;
        }
        __syncthreads();

        // Randomize size and position for the next level
        x += hiprand(&state) % (width - size);
        y += hiprand(&state) % (height - size);
        size = size / 2;
        if (size < 1) size = 1;

        __syncthreads();
    }

    // Copy results to global memory
    if (threadIdx.x == 0) {
        for (int i = 0; i < localIndex; ++i) {
            nodes[i] = localNodes[i];
        }
    }
}

void launchRandomQuadtreeKernel(uchar4* devPtr, int width, int height, int maxDepth, unsigned long long seed) {
    // Allocate memory for the quadtree nodes
    int maxNodes = (1 << (2 * maxDepth)) - 1;
    QuadTreeNode* nodes;
    hipMalloc(&nodes, maxNodes * sizeof(QuadTreeNode));

    // Generate the random quadtree
    generateRandomQuadtree << <1, 1 >> > (nodes, width, height, maxDepth, seed);
    hipDeviceSynchronize();

    // Draw the quadtree
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    drawQuadtree << <gridSize, blockSize >> > (devPtr, width, height, nodes, maxNodes);
    hipDeviceSynchronize();

    // Free the quadtree node memory
    hipFree(nodes);
}
